
#include <hip/hip_runtime.h>
// REQUIRES: !system-solaris

// The test cannot be run on Darwin because /Users will be treated as a MSVC option.

// RUN: %clang_cl -### -target x86_64-pc-windows-msvc --offload-arch=sm_35 -fgpu-rdc \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN:   /Wall -x cuda -- %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix=CUDA

// RUN: %clang_cl -### -target x86_64-pc-windows-msvc --offload-arch=gfx1010 -fgpu-rdc --hip-link \
// RUN:   --rocm-path=%S/Inputs/rocm /Wall -x hip -- %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix=HIP

// CUDA: "-cc1" "-triple" "nvptx64-nvidia-cuda" "-aux-triple" "x86_64-pc-windows-msvc"
// INTEL CUDA-SAME: "-Wall"
// CUDA: ptxas
// CUDA: "-cc1" "-triple" "x86_64-pc-windows-msvc{{.*}}" "-aux-triple" "nvptx64-nvidia-cuda"
// INTEL CUDA-SAME: "-Wall"
// CUDA: link

// HIP: "-cc1" "-triple" "x86_64-pc-windows-msvc{{.*}}" "-aux-triple" "amdgcn-amd-amdhsa"
// INTEL HIP-SAME: "-Wall"
// HIP: "-cc1" "-triple" "amdgcn-amd-amdhsa" "-aux-triple" "x86_64-pc-windows-msvc"
// INTEL HIP-SAME: "-Wall"
// HIP: {{lld.* "-flavor" "gnu" "-m" "elf64_amdgpu"}}
// HIP: {{link.* "amdhip64.lib"}}

// CMake uses this option when finding packages for HIP, so
// make sure it does not cause error.

// RUN: %clang_cl --print-libgcc-file-name
